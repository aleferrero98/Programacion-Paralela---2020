#include "stdio.h"
#include "float.h"
#include <hip/hip_runtime.h>

#define HILOS 128
#define PATH "./inputs/randomData_2M_3feature.csv"
#define CANT_FEATURES 3
#define CANT_MEANS 4
#define CANT_ITERACIONES 100
#define MAX_DOUBLE DBL_MAX

//Funciones CUDA
__global__ void kMeansClusterAssignment(double* means_dev, double* items_dev, int *clusterAsignado_dev,int *countChangeItem_dev );
__global__ void kMeansCentroidUpdate(double *items_dev, int *clusterAsignado_dev, double *means_dev, int *d_clust_sizes);
__device__ u_int64_t Classify(double* means_dev, double* item, int cant_means, int cant_features);
__device__ double distanciaEuclidiana(double* x , double* y, int length);

//Funciones HOST
double** CalculateMeans(double* items_dev, double** means, u_int64_t size_lines, int *clusterAsignado_dev, int nBloques, int hilosB);
double*** FindClusters(int *clusterAsignado_dev, u_int64_t cant_items, double **items);
u_int64_t CalcLines(char filename[50]);
double **alloc_2d_double(u_int64_t rows, u_int64_t cols);
double** ReadData(char filename[50], u_int64_t size_lines, u_int8_t cant_features);
void searchMinMax(double** items, u_int64_t size_lines, double* minimo, double* maximo, u_int8_t cant_features);
double** InitializeMeans(u_int16_t cant_means, double* cMin, double* cMax, u_int8_t cant_features);
__host__ void check_CUDA_Error(const char *mensaje);

//Constantes de CUDA
__constant__ u_int64_t CANT_ITEMS_CUDA;

int main()
{
    //Declaracion de eventos para tomar tiempos
    hipEvent_t start;
    hipEvent_t stop;

    //Creacion de eventos
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Marca de inicio CalcLines y ReadData
    hipEventRecord(start,0);
    //Calcula la cantidad de lineas del CSV
    u_int64_t size_lines = CalcLines(PATH);

    hipMemcpyToSymbol(HIP_SYMBOL(CANT_ITEMS_CUDA), &size_lines, sizeof(u_int64_t));
    check_CUDA_Error("ERROR en hipMemcpyToSymbol");

    // double maxDouble = DBL_MAX;
    // cudaMemcpyToSymbol(MAX_DOUBLE, &maxDouble, sizeof(double));
    // check_CUDA_Error("ERROR en cudaMemcpyToSymbol");

    double **items = ReadData(PATH, size_lines, CANT_FEATURES);
    //Marca de final CalcLines y ReadData
    hipEventRecord(stop,0);
    //Sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    //Calculo del tiempo en milisegundos
    float elapsedTime2;
    hipEventElapsedTime(&elapsedTime2,start,stop);

    //Marca de inicio SearchMinMax, Calculo de hilos-bloques CUDA e Inicializacion Medias
    hipEventRecord(start,0);
    double *cMin, *cMax;
    cMin = (double*) malloc(CANT_FEATURES * sizeof(double));
    cMax = (double*) malloc(CANT_FEATURES * sizeof(double));
    //Encuentra el minimo y maximo de cada columna (o feature)
    searchMinMax(items, size_lines, cMin, cMax, CANT_FEATURES);

    printf("MIN: %lf, MAX: %lf\n", cMin[0], cMax[0]);
    
    // calculamos el numero de bloques necesario para un tamaño de bloque fijo
    int nBloques = size_lines/HILOS;
    if (size_lines%HILOS != 0)
    {
        nBloques = nBloques + 1;
    }
    int hilosB = HILOS;

    //Inicializa las means (medias) con valores estimativos
    double** means = InitializeMeans(CANT_MEANS, cMin, cMax, CANT_FEATURES);

    //Marca de final SearchMinMax, Calculo de hilos-bloques CUDA e Inicializacion Medias
    hipEventRecord(stop,0);
    //Sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    //Calculo del tiempo en milisegundos
    float elapsedTime3;
    hipEventElapsedTime(&elapsedTime3,start,stop);

    //Almacena los indices de los items
    int *clusterAsignado_dev = 0;
    hipMalloc(&clusterAsignado_dev,size_lines*sizeof(int));
    hipMemset(clusterAsignado_dev,0,size_lines*sizeof(int));

    double* items_dev;
    hipMalloc( (void**)&items_dev, size_lines*CANT_FEATURES*sizeof(double));
    check_CUDA_Error("ERROR en hipMalloc");
    hipMemcpy( items_dev, &items[0][0], size_lines*CANT_FEATURES*sizeof(double), hipMemcpyHostToDevice );
    check_CUDA_Error("ERROR en hipMemcpy items_dev");
    
    //Marca de inicio CalculateMeans
    hipEventRecord(start,0);
    //Funcion que calcula las medias nuevas
    means = CalculateMeans(items_dev, means, size_lines, clusterAsignado_dev ,nBloques, hilosB);
    //Marca de final CalculateMeans
    hipEventRecord(stop,0);
    //Sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    //Calculo del tiempo en milisegundos
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    //Marca de inicio FindCluster
    hipEventRecord(start,0);
    //Funcion que calcula las medias nuevas
    double ***clusters = FindClusters(clusterAsignado_dev, size_lines, items);
    //Marca de final CalculateMeans
    hipEventRecord(stop,0);
    //Sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    //Calculo del tiempo en milisegundos
    float elapsedTime4;
    hipEventElapsedTime(&elapsedTime4,start,stop);

    //Liberacion de recursos
    for(int n = 0; n < CANT_MEANS; n++){
        for(u_int64_t m = 0; m < size_lines; m++){
            free(clusters[n][m]);
        } 
        free(clusters[n]);
    }
    free(clusters);
    free(items[0]);
    free(items);
    free(means[0]);
    free(means);
    free(cMin);
    free(cMax);
    hipFree(clusterAsignado_dev);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    //Impresion de resultados
    printf("> Tiempo de ejecucion de CalcLines y ReadData: %f ms\n",elapsedTime2);
    printf("> Tiempo de ejecucion de SearchMinMax, Calculo de hilos-bloques CUDA e Inicializacion Medias: %f ms\n",elapsedTime3);
    printf("> Tiempo de ejecucion de CalculateMeans: %f ms\n",elapsedTime);
    printf("> Tiempo de ejecucion de FindCluster: %f ms\n",elapsedTime4);
    printf("> Tiempo de total del programa: %f ms\n", elapsedTime + elapsedTime2 + elapsedTime3 + elapsedTime4);

    return EXIT_SUCCESS;
}
/**
 * @brief Funcion que se encarga de armar una matriz 3D, donde se insertaran los items de acuerdo a su clasificacion
 * @param clusterAsignado_dev Arreglo 1D del cluster a que corresponde cada item
 * @param cant_items Cantidad de items
 * @param items Items a clasificar
 * @return Arreglo 3D de Clusters finales de acuerdo a la clasificacion de los items en cada media
 */
double*** FindClusters(int *clusterAsignado_dev, u_int64_t cant_items, double **items)
{
    // clusters es un array de 3 dimensiones, es un conjunto de clusters.
    // cada cluster es un conjunto de items.
    // cada item es un conjunto de features.
    double ***clusters = (double ***) malloc(CANT_MEANS * sizeof(double**));

    //Inicializa clusters
    for(u_int8_t n = 0; n < CANT_MEANS; n++){
        clusters[n] = (double **) malloc(cant_items * sizeof(double*));
        for(u_int64_t m = 0; m < cant_items; m++){
            clusters[n][m] = (double *) malloc(CANT_FEATURES * sizeof(double));
        }
    }

    int *clusterAsignado = (int*)malloc(cant_items*sizeof(int));
    hipMemcpy(clusterAsignado, clusterAsignado_dev, cant_items*sizeof(int), hipMemcpyDeviceToHost );
    int indices_series[CANT_MEANS]; 
    memset(indices_series, 0, sizeof(int)*CANT_MEANS);
    for(u_int64_t i = 0; i < cant_items; i++){
        for(u_int8_t j = 0; j < CANT_FEATURES; j++){ //se cargan todas las features del item al cluster
            clusters[clusterAsignado[i]][indices_series[clusterAsignado[i]]][j] = items[i][j];
        }
        indices_series[clusterAsignado[i]]++;
    }
    return clusters;
}
/**
 * @brief Funcion que se encarga de clasificar los items en las medias correspondientes
 * @param items_dev Items a clasificar, cada item contiene un valor por Feature, representada como arreglo 1D
 * @param means_dev Matriz de medias (Cantidad de Features * Cantidad de Medias), representada como arreglo 1D
 * @param size_lines Cantidad de items
 * @param clusterAsignado_dev Arreglo 1D del cluster a que corresponde cada item
 * @param nBloques Cantidad de bloques CUDA
 * @param hilosB Cantidad de hilos CUDA
 * @return Arreglo 2D de Medias finales de acuerdo a la clasificacion de los items
 */
double** CalculateMeans(double* items_dev, double** means, u_int64_t size_lines, int *clusterAsignado_dev, int nBloques, int hilosB)
{
    double minPorcentaje;
    //define el porcentaje minimo de cambio de items entre clusters para que continue la ejecucion del algoritmo
    minPorcentaje = 0.001 * (double) size_lines;

    printf("Porentaje minimo = %.2lf\n", minPorcentaje);

    double* means_dev;
    hipMalloc( (void**)&means_dev, CANT_MEANS*CANT_FEATURES*sizeof(double));
    check_CUDA_Error("ERROR en hipMalloc");

    /*Arreglo de cluster sizes*/
    //Creo y reseteo a 0 la variable de host
    int *h_clust_sizes = (int*)malloc(CANT_MEANS*sizeof(int));
    memset(h_clust_sizes, 0, sizeof(int)*CANT_MEANS);
    //cudaMemset(countChangeItem_dev, 0, sizeof(int));
    //Creo la variable de device
    int *d_clust_sizes = 0;
    hipMalloc(&d_clust_sizes,CANT_MEANS*sizeof(float));
    check_CUDA_Error("ERROR en hipMalloc d_clust_sizes ");
    //Copio lo que hay en host a device
    hipMemcpy(d_clust_sizes,h_clust_sizes,CANT_MEANS*sizeof(int),hipMemcpyHostToDevice);
    check_CUDA_Error("ERROR en hipMemcpy d_clust_sizes ");
    //Almacena contador de cambios de items
    int *countChangeItem_dev = 0;
    hipMalloc(&countChangeItem_dev,sizeof(int));
    
    int *countChangeItem = (int*)malloc(sizeof(int));

    //Calcula las medias
    for(int j = 0; j < CANT_ITERACIONES; j++) {
        
        printf("Iteracion: %d\n", j);

        //En cada iteracion, cantidad de cambios es 0
        //memset(countChangeItem, 0, sizeof(int));
                
        //Paso lo que hay en means a la placa luego de cambiarlo
        hipMemcpy( means_dev, &means[0][0], CANT_MEANS*CANT_FEATURES*sizeof(double), hipMemcpyHostToDevice );
        check_CUDA_Error("ERROR en hipMemcpy means_dev");

        //Reseteo la cantidad de elementos de cada media en cada iteracion
        hipMemset(d_clust_sizes,0,CANT_MEANS*sizeof(int));
        check_CUDA_Error("ERROR en hipMemset means_dev");

        kMeansClusterAssignment<<<nBloques,hilosB>>>(items_dev, means_dev, clusterAsignado_dev, countChangeItem_dev);

        //Copio las nuevas medias obtenidas en la placa a las medias de Host
        hipMemcpy(countChangeItem,countChangeItem_dev,sizeof(int),hipMemcpyDeviceToHost);

        //Reseteo means para la placa, ya que se va a cambiar
        hipMemset(means_dev,0,CANT_MEANS*CANT_FEATURES*sizeof(double));
        check_CUDA_Error("ERROR en hipMemset means_dev");

        kMeansCentroidUpdate<<<nBloques,hilosB>>>(items_dev,clusterAsignado_dev,means_dev,d_clust_sizes);

        //Copio las nuevas medias obtenidas en la placa a las medias de Host
        hipMemcpy(&means[0][0],means_dev,CANT_MEANS*CANT_FEATURES*sizeof(double),hipMemcpyDeviceToHost);
        check_CUDA_Error("ERROR en hipMemcpy means_dev 3");
        //Copio la cantidad de items de cada medias obtenidas en la placa al arreglo del host
        hipMemcpy(h_clust_sizes, d_clust_sizes, CANT_MEANS*sizeof(int), hipMemcpyDeviceToHost );
        check_CUDA_Error("ERROR en hipMemcpy h_clust_sizes ");

        for (int a = 0; a < CANT_MEANS; a++)
        {
            for(int b=0; b < CANT_FEATURES; b++)
            {
                //Asigno el nuevo valor de las medias sacando promedio
                means[a][b] = means[a][b] / h_clust_sizes[a];
            }
            printf("Mean[%d] -> (%lf,%lf,%lf)\n", a, means[a][0], means[a][1],  means[a][2]);
            printf("Cluster[%d] -> %d\n", a, h_clust_sizes[a]);
        }
        
        //Comparo la cantidad de items cambiado en la iteracion actual con la anterior y si es menor al porcentaje
        //se deja de iterar
        printf("Cant cambios: %d\n",*countChangeItem);
        if(*countChangeItem < minPorcentaje){break;}
        //Reseteo cantidad de camios para la placa, ya que se va a cambiar
        hipMemset(countChangeItem_dev,0,sizeof(int));
    }

    hipFree(items_dev);
    hipFree(means_dev);
    hipFree(d_clust_sizes);
    free(h_clust_sizes);
    hipFree(countChangeItem_dev);
    free(countChangeItem);
    return means;
}
/**
 * @brief Funcion que se encarga de obtener las sumas en cada media y la cantidad de elementos
 * @param items_dev Items a clasificar, cada item contiene un valor por Feature, representada como arreglo 1D
 * @param clusterAsignado_dev Arreglo 1D del cluster a que corresponde cada item
 * @param means_dev Matriz de medias (Cantidad de Features * Cantidad de Medias), representada como arreglo 1D
 * @param d_clust_sizes Arreglo 1D de la cantidad de items de cada media del cluster
 */
__global__ void kMeansCentroidUpdate(double *items_dev, int *clusterAsignado_dev, double *means_dev, int *d_clust_sizes)
{

	//Obtengo el ID de cada hilo
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//Elimino aquellos que no deban trabajar
	if (idx >= CANT_ITEMS_CUDA) return;

	//Obtengo el ID de los hilos a nivel de bloque
	const int s_idx = threadIdx.x;

    //Armo un arreglo de items para cada bloque en memoria compartida
    __shared__ double items_bloque[HILOS][CANT_FEATURES];

    for(int i = 0; i < CANT_FEATURES; i++){
        items_bloque[s_idx][i] = items_dev[idx*CANT_FEATURES + i];
    }

    //Armo un arreglo de los cluster asignados para cada bloque en memoria compartida
	__shared__ int clusterAsignado_bloque[HILOS];
    clusterAsignado_bloque[s_idx] = clusterAsignado_dev[idx];

	__syncthreads();

    //Si es el hilo 0 de cada bloque, entonces suma los valores dentro de los arreglo compartido
	if(s_idx==0)
	{
        int limite = ((idx + blockDim.x) < CANT_ITEMS_CUDA)? blockDim.x : (CANT_ITEMS_CUDA - idx);

        //Creo arreglos de suma de valores del cluster del bloque y la cantidad de items de cada media
		double clust_sums[CANT_MEANS][CANT_FEATURES]={{0},{0},{0},{0}};
        int clust_sizes[CANT_MEANS]={0};

        //Se recorre el bloque, incrementando el cluster sizes de acuerdo a la media asignada y lo sumo 
		for(int j=0; j < limite; ++j)
		{
            int clust_id = clusterAsignado_bloque[j];
            clust_sizes[clust_id]+=1;
            for(int k = 0; k < CANT_FEATURES; ++k)
            {
                clust_sums[clust_id][k]+=items_bloque[j][k];
            }
		}

        //Por ultimo agregamos de forma atomica al arreglo means_dev la suma de todos los items designados en cada cluster
        //y al arreglo d_clust_sizes la cantidad de items en cada media
        int indice;
		for(int z=0; z < CANT_MEANS; ++z)
		{
            indice = z*CANT_FEATURES;
            for(int s=0; s < CANT_FEATURES ; s++)
            {
                atomicAdd(&means_dev[indice+s],clust_sums[z][s]);
            }
            atomicAdd(&d_clust_sizes[z],clust_sizes[z]);
        }
	}

	__syncthreads();
}
/**
 * @brief Funcion que se encarga de asignar los indices de cluster a cada item 
 * @param items_dev Items a clasificar, cada item contiene un valor por Feature, representada como arreglo 1D
 * @param means_dev Matriz de medias (Cantidad de Features * Cantidad de Medias), representada como arreglo 1D
 * @param clusterAsignado_dev Arreglo 1D del cluster a que corresponde cada item
 */
__global__ void kMeansClusterAssignment(double *items_dev, double *means_dev, int *clusterAsignado_dev,int *countChangeItem_dev )
{
    
    //Obtengo el ID para cada hilo
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    //Descarto aquellos hilos que no deban trabajar
	if (idx >= CANT_ITEMS_CUDA) return;
    
    //Obtengo el item correspondiente a cada hilo
    double *item = &items_dev[idx*CANT_FEATURES];

    u_int64_t index = Classify(means_dev, item, CANT_MEANS, CANT_FEATURES);

    if(clusterAsignado_dev[idx] != (int)index)
    {
        atomicAdd(countChangeItem_dev,1);
    }
    //Asigno cada item en un cluster y almaceno el indice de clasificacion en un arreglo
	clusterAsignado_dev[idx]=(int)index;
}
/**
 * @brief Funcion que se encarga de obtener el indice del cluster al que pertenece el item
 * @param means_dev Matriz de medias (Cantidad de Features * Cantidad de Medias), representada como arreglo 1D
 * @param item Item a clasificar
 * @param cant_means Cantidad de Medias
 * @param cant_features Cantidad de Features
 * @return Indice del cluster al que corresponde el Item
 */
__device__ u_int64_t Classify(double* means_dev, double* item, int cant_means, int cant_features){
    double minimun = MAX_DOUBLE;
    int index = -1;
    double distance;

    for(int i = 0; i < cant_means; i++){
        //calcula la distancia de un item a la media
        //printf("Means_dev: %ld\n", means_dev[i*3]);
        distance = distanciaEuclidiana(item, &means_dev[i*cant_features], cant_features);
        if(distance < minimun){
            minimun = distance;
            index = i;
        }
    }
    return (u_int64_t) index;
}
/**
 * @brief Funcion que se encarga de calcular la distancia Euclideana entre el item y las distintas Medias (2 vectores)
 * @param x Item (Vector 1)
 * @param y Medias (Vector 2)
 * @param length longitud del vector (Cantidad de Features)
 * @return Distancia euclidiana entre ambos vectores.
 */
__device__ double distanciaEuclidiana(double* x , double* y, int length){
    double distancia = 0;
    for(int i = 0; i < length; i++){
        distancia += pow((x[i] - y[i]), 2);
    }
    return sqrt(distancia);
}
/**
 * @brief Funcion que se encarga de calcular la cantidad de items a clasificar
 * @param filename nombre del archivo
 * @return cantidad de lineas (o items) del archivo
 */
u_int64_t CalcLines(char filename[50]) {
    printf(filename);
    FILE *f = fopen(filename, "r");
    u_int64_t cant_lines = 0; 
    char* cadena = (char*) calloc(100, sizeof(char));
    char* valor;
    while(fgets(cadena, 100, f)){
        valor = strstr(cadena, ",");
        valor++;
        if(valor != NULL && strcmp(valor,"values\n") && strcmp(valor,"\n")){
            cant_lines++;
        }
    }
    free (cadena);
    fclose(f);
    printf("Cantidad de items: %ld\n", cant_lines);

    return cant_lines;
}
/**
 * @brief Funcion que se encarga allocar una matriz 2D
 * @param rows filas de la matriz
 * @param cols columnas de la matriz
 * @return Matriz 2D
 */
double **alloc_2d_double(u_int64_t rows, u_int64_t cols) {
    double *data = (double *)malloc(rows * cols * sizeof(double));
    double **array= (double **)malloc(rows * sizeof(double*));
    for (u_int64_t i = 0; i < rows; i++)
        array[i] = &(data[cols*i]);

    return array;
}
/**
 * @brief Busca el minimo y maximo valor para cada feature del arreglo items.
 * @param items datos a clasificar
 * @param size_lines cantidad de items
 * @param minimo arreglo de los valores minimos de cada feature
 * @param maximo arreglo de los valores maximos de cada feature
 * @param cant_features cantidad de caracteristicas que tiene cada item
 */
void searchMinMax(double** items, u_int64_t size_lines, double* minimo, double* maximo, u_int8_t cant_features){

    //Define el maximo como el minimo valor de tipo DOUBLE y el minimo como el maximo valor de tipo DOUBLE
    for(int n = 0; n < cant_features; n++){
        maximo[n] = DBL_MIN;
        minimo[n] = DBL_MAX;
    }
    
    for(u_int64_t i = 0; i < size_lines; i++){  //recorremos cada item
        for(u_int8_t j = 0; j < cant_features; j++){  //recorremos cada feature
            if(items[i][j] < minimo[j]){
                minimo[j] = items[i][j];
            }
            if(items[i][j] > maximo[j]){
                maximo[j] = items[i][j];
            }
        }
    }

    printf("maximos: %lf, %lf, %lf\n", maximo[0], maximo[1], maximo[2]);
    printf("minimos: %lf, %lf, %lf\n", minimo[0], minimo[1], minimo[2]);
} 
/**
 * @brief Lee el archivo indicado y carga el arreglo de items.
 * @param filename string nombre del archivo que contiene los datos
 * @param size_lines cantidad de lineas del archivo
 * @param cant_features cantidad de features de cada item (cantidad de columnas del archivo separadas por comas) 
 * @return arreglo doble con cantidad de filas igual a cantidad de items y cantidad de columnas igual a cantidad de features.
 */
double** ReadData(char filename[50], u_int64_t size_lines, u_int8_t cant_features){
    
    FILE *file = fopen(filename, "r");
    rewind(file);

    //Definimos un arreglo de arreglos (cada item consta de 2 o mas features)
    double** items = (double **) alloc_2d_double(size_lines, cant_features);

    char* line = (char*)calloc(100, sizeof(char));
    double feature;
    u_int64_t i = 0, j = 0;
    char* ptr;

    while(fgets(line, 100, file)){
        j = 0;
        char *item = strstr(line, ","); //se ignora el primer elemento del archivo (indice)
        item++;
        if(item != NULL && strcmp(item, "values\n") && strcmp(item, "\n")){ //Para recortar la cadena y tomar solo el segundo dato
           // item[strlen(item)-1] = '\0';
            char *token = strtok(item, ","); //separa los elementos de la linea por comas
            while(token != NULL){
                feature = strtod(token, &ptr); //Pasaje a double
                items[i][j] = feature; //Almacenamiento en item
                j++;
                token = strtok(NULL, ","); //busco el siguiente token
            }
            i++;
        }
    }
    free(line);
    fclose(file);

    return items;
}
/**
 * @brief Funcion que se encarga de detectar error de CUDA 
 * @param mensaje Mensaje de error CUDA
 */
__host__ void check_CUDA_Error(const char *mensaje)
{
    hipError_t error;
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
        printf("\npulsa INTRO para finalizar...");
        fflush(stdin);
        char tecla = getchar();
        exit(-1);
    }
}
/**
 * @brief Inicializa el arreglo de medias en valores equiespaciados en el rango de datos.
 * @param cant_means cantidad de medias o clusters
 * @param cMin vector con los valores minimos de cada feature
 * @param cMax vector con los valores maximos de cada feature
 * @param cant_features cantidad de features (o columnas) de cada item
 * @return arreglo con las medias (1 por cada cluster).
 * Ejemplo: range: 20 (0 a 19)
 *          cantMeans -> 4
 *          jump: 20 / 4 = 5
 *          means[0] = 0 + 0.5 * 5 = 2.5
 *          means[1] = 0 + 1.5 * 5 = 7.5
 *          means[2] = 0 + 2.5 * 5 = 12.5
 *          means[3] = 0 + 3.5 * 5 = 17.5
 */
double** InitializeMeans(u_int16_t cant_means, double* cMin, double* cMax, u_int8_t cant_features){
    /*        |__Feature 0__|__Feature 1__|__Feature 2__|                
        Media0|_____________|_____________|_____________|
        Media1|_____________|_____________|_____________|
    */
    double **means = (double **) alloc_2d_double(cant_means, cant_features);
    
    //definimos el salto de un valor de media al siguiente
    double *jump = (double *) malloc(cant_features * sizeof(double));
    for(u_int8_t n = 0; n < cant_features; n++){
        jump[n] = (double) (cMax[n] - cMin[n]) / cant_means;
    }

   printf("\nValores de las medias iniciales:\n");
    for(u_int16_t i = 0; i < cant_means; i++){
        for(u_int8_t j = 0; j < cant_features; j++){
            means[i][j] = cMin[j] + (0.5 + i) * jump[j];
        }
        printf("Mean[%d] -> (%lf,%lf,%lf)\n", i, means[i][0], means[i][1],  means[i][2]);
    }
    free(jump);
    return means;
}